
#include <hip/hip_runtime.h>
#include <cstdint>
#include <memory.h>
#include <cstdio>
#include <ctime>
#include <thread>
#include <vector>
#include <mutex>
#include <chrono>
#include <string>

#define RANDOM_MULTIPLIER 0x5DEECE66DULL
#define RANDOM_ADDEND 0xBULL
#define RANDOM_MASK ((1ULL << 48ULL) - 1ULL)

#define CHUNK_SEED_BOTTOM_4 (CHUNK_SEED & 0xFULL)
#define CHUNK_SEED_BIT_5 ((CHUNK_SEED >> 4ULL) & 1ULL)

#ifndef FLOOR_LEVEL
#define FLOOR_LEVEL 63LL
#endif

#ifndef WANTED_CACTUS_HEIGHT
#define WANTED_CACTUS_HEIGHT 20LL
#endif

#ifndef WORK_UNIT_SIZE
#define WORK_UNIT_SIZE (1ULL << 23ULL)
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256ULL
#endif

#ifndef GPU_COUNT
#define GPU_COUNT 1ULL
#endif

#ifndef OFFSET
#define OFFSET 0
#endif

#ifndef END
#define END (1ULL << 48ULL)
#endif

#ifndef CHUNK_SEED
#define CHUNK_SEED 9567961692053ULL
#endif

#ifndef NEIGHBOR1
#define NEIGHBOR1 856ULL
#endif

#ifndef NEIGHBOR2
#define NEIGHBOR2 344ULL
#endif

#ifndef NEIGHBOR3
#define NEIGHBOR3 840ULL
#endif

#ifndef DIAGONAL_INDEX
#define DIAGONAL_INDEX 0ULL
#endif

#ifndef CACTUS_HEIGHT
#define CACTUS_HEIGHT 12ULL
#endif

namespace java_random {

    // Random::next(bits)
    __device__ inline uint32_t next(uint64_t *random, int32_t bits) {
        *random = (*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
        return (uint32_t) (*random >> (48ULL - bits));
    }

    __device__ inline int32_t next_int_unknown(uint64_t *seed, int16_t bound) {
        if ((bound & -bound) == bound) {
            *seed = (*seed * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
            return (int32_t) ((bound * (*seed >> 17ULL)) >> 31ULL);
        }

        int32_t bits, value;
        do {
            *seed = (*seed * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
            bits = *seed >> 17ULL;
            value = bits % bound;
        } while (bits - value + (bound - 1) < 0);
        return value;
    }

    // Random::nextInt(bound)
    __device__ inline uint32_t next_int(uint64_t *random) {
        return java_random::next(random, 31) % 3;
    }

}

__global__ __launch_bounds__(BLOCK_SIZE, 2) void crack(uint64_t seed_offset, int32_t *num_seeds, uint64_t *seeds) {
    uint64_t originalSeed = ((blockIdx.x * blockDim.x + threadIdx.x + seed_offset) << 4ULL) | CHUNK_SEED_BOTTOM_4;
    uint64_t seed = originalSeed;

    int8_t heightMap[1024];

#pragma unroll
    for (int i = 0; i < 1024; i++) {
        heightMap[i] = FLOOR_LEVEL;
    }

    int32_t currentHighestPos = 0, posMap;
    int16_t initialPosX, initialPosY, initialPosZ, initialPos;
    int16_t posX, posY, posZ;

    int16_t i, a, j;
    int8_t position = -1;

    for (i = -90; i < 0; i += 9) {
        // Keep, most threads finish early this way
        if (heightMap[currentHighestPos] - WANTED_CACTUS_HEIGHT - FLOOR_LEVEL < i)
            return;

        initialPosX = java_random::next(&seed, 4) + 8;
        initialPosZ = java_random::next(&seed, 4) + 8;

        initialPos = initialPosX + initialPosZ * 32;

        if (position == -1) {
            if (initialPos == NEIGHBOR1) {
                position = 0;
            } else if (initialPos == NEIGHBOR2) {
                position = 1;
            } else if (initialPos == NEIGHBOR3) {
                position = 2;
            }

            if (position != -1) {
                uint64_t bit = (originalSeed >> 4ULL) & 1ULL;

                if (position != DIAGONAL_INDEX) {
                    if (bit == CHUNK_SEED_BIT_5) return;
                } else {
                    if (bit != CHUNK_SEED_BIT_5) return;
                }

                heightMap[initialPos] += CACTUS_HEIGHT;

                if (heightMap[currentHighestPos] < heightMap[initialPos]) {
                    currentHighestPos = initialPos;
                }
            }
        }

        initialPosY = java_random::next_int_unknown(&seed, (heightMap[initialPosX + initialPosZ * 32] + 1) * 2);

        for (a = 0; a < 10; a++) {
            posX = initialPosX + java_random::next(&seed, 3) - java_random::next(&seed, 3);
            posY = initialPosY + java_random::next(&seed, 2) - java_random::next(&seed, 2);
            posZ = initialPosZ + java_random::next(&seed, 3) - java_random::next(&seed, 3);

            posMap = posX + posZ * 32;

            if (position == -1) {
                if (posMap == NEIGHBOR1) {
                    position = 0;
                } else if (posMap == NEIGHBOR2) {
                    position = 1;
                } else if (posMap == NEIGHBOR3) {
                    position = 2;
                }

                if (position != -1) {
                    uint64_t bit = (originalSeed >> 4ULL) & 1ULL;

                    if (position != DIAGONAL_INDEX) {
                        if (bit == CHUNK_SEED_BIT_5) return;
                    } else {
                        if (bit != CHUNK_SEED_BIT_5) return;
                    }

                    heightMap[posMap] += CACTUS_HEIGHT;

                    if (heightMap[currentHighestPos] < heightMap[posMap]) {
                        currentHighestPos = posMap;
                    }
                }
            }

            // Keep
            if (posY <= heightMap[posMap])
                continue;

            for (j = 0; j < 1 + java_random::next_int_unknown(&seed, java_random::next_int(&seed) + 1); j++) {
                if ((posY + j - 1) > heightMap[posMap] || posY < 0) continue;
                if ((posY + j) <= heightMap[(posX + 1) + posZ * 32]) continue;
                if ((posY + j) <= heightMap[posX + (posZ - 1) * 32]) continue;
                if ((posY + j) <= heightMap[(posX - 1) + posZ * 32]) continue;
                if ((posY + j) <= heightMap[posX + (posZ + 1) * 32]) continue;

                heightMap[posMap]++;

                if (heightMap[currentHighestPos] < heightMap[posMap]) {
                    currentHighestPos = posMap;
                }
            }
        }

        if (heightMap[currentHighestPos] - FLOOR_LEVEL >= WANTED_CACTUS_HEIGHT) {
            uint64_t addend = 0;
            if (position == 0)
                addend = NEIGHBOR1;
            if (position == 1)
                addend = NEIGHBOR2;
            if (position == 2)
                addend = NEIGHBOR3;
            seeds[atomicAdd(num_seeds, 1)] = (addend << 48ULL) | originalSeed;
            return;
        }
    }
}

struct GPU_Node {
    int* num_seeds;
    uint64_t* seeds;
};

void setup_gpu_node(GPU_Node* node, int32_t gpu) {
    hipSetDevice(gpu);
    hipMallocManaged(&node->num_seeds, sizeof(*node->num_seeds));
    hipMallocManaged(&node->seeds, 1ULL << 10ULL); // approx 1kb
}

GPU_Node nodes[GPU_COUNT];
uint64_t offset = OFFSET;
uint64_t count = 0;
std::mutex info_lock;

void gpu_manager(int32_t gpu_index) {
    std::string fileName = "kaktoos_seeds" + std::to_string(gpu_index) + ".txt";
    FILE *out_file = fopen(fileName.c_str(), "a");
    hipSetDevice(gpu_index);
    while (offset < END) {
        *nodes[gpu_index].num_seeds = 0;
        crack<<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE, 0>>> (offset, nodes[gpu_index].num_seeds, nodes[gpu_index].seeds);
        info_lock.lock();
        offset += WORK_UNIT_SIZE;
        info_lock.unlock();
        hipDeviceSynchronize();
        for (int32_t i = 0, e = *nodes[gpu_index].num_seeds; i < e; i++) {
            fprintf(out_file, "%lld\n", (long long int)nodes[gpu_index].seeds[i]);
            printf("Found seed: %lld\n", (long long int)nodes[gpu_index].seeds[i]);
        }
        fflush(out_file);
        info_lock.lock();
        count += *nodes[gpu_index].num_seeds;
        info_lock.unlock();
    }
    fclose(out_file);
}

int main() {
    printf("Searching %ld total seeds...\n", END - OFFSET);

    std::thread threads[GPU_COUNT];

    time_t startTime = time(nullptr), currentTime;
    for(int32_t i = 0; i < GPU_COUNT; i++) {
        setup_gpu_node(&nodes[i], i);
        threads[i] = std::thread(gpu_manager, i);
    }

    using namespace std::chrono_literals;

    while (offset < END) {
        time(&currentTime);
        int timeElapsed = (int)(currentTime - startTime);
        double speed = (double)(offset - OFFSET) / (double)timeElapsed / 1000000.0;
        printf("Searched %lld seeds, offset: %lld found %lld matches. Time elapsed: %ds. Speed: %.2fm seeds/s. %f%%\n",
               (long long int)(offset - OFFSET),
               (long long int)offset,
               (long long int)count,
               timeElapsed,
               speed,
               (double)(offset - OFFSET) / (END - OFFSET) * 100);

        std::this_thread::sleep_for(0.5s);
    }

    for (auto &thread : threads) {
        thread.join();
    }

    printf("Done!\n");
    printf("But, verily, it be the nature of dreams to end.\n");

}